#include "hip/hip_runtime.h"
/*
 * CUDARadiosity 0.87a
 * Copyright 2012 by David Sargeant, Patrick Gradie, Michael Rogers
 * 
 * Based on code from rrv (Radiosity Renderer and Visualizer) by TODO
 * Distributed under GPL (see <http://www.gnu.org/licenses/>)
 * 
 */

#include "FormFactorEngine.h"
#include "TriangleSet.h"
#include "PatchRandomAccessEnumerator.h"
#include "PatchCacheLine.h"
#include <cuPrintf.cu>

#include <GL/glu.h>

#define EPSILON 0.000001

#if defined(__WIN32__) || defined(_WIN32) || defined(__CYGWIN__)
	#include <windows.h>
	#include <wingdi.h>
	#include <GL/glext.h>
#else
	#include <X11/Xlib.h>
	#include <GL/glx.h>
#endif

#include <math.h>
#ifndef M_PI
	#define M_PI		3.14159265358979323846
#endif
#ifndef EDGE_L1
	#define EDGE_L1 FormFactorEngine::EDGE_LENGTH
#endif
#ifndef SCREEN_SIZE
	#define SCREEN_SIZE EDGE_L1*EDGE_L1*3
#endif

//#include <cutil.h>
//#include <cutil_math.h>
//#include <shrUtils.h>
#include <iostream>
#include <string>
//#include <book.h>

using namespace std;

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

//typedef struct {
//	int edge;
//	double* elements;
//} Matrix;

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


#define HANDLE_NULL( a ) {if (a == NULL) { \
                            printf( "Host memory failed in %s at line %d\n", \
                                    __FILE__, __LINE__ ); \
                            exit( EXIT_FAILURE );}}

template< typename T >
void swap( T& a, T& b ) {
    T t = a;
    a = b;
    b = t;
}


void* big_random_block( int size ) {
    unsigned char *data = (unsigned char*)malloc( size );
    HANDLE_NULL( data );
    for (int i=0; i<size; i++)
        data[i] = rand();

    return data;
}

int* big_random_block_int( int size ) {
    int *data = (int*)malloc( size * sizeof(int) );
    HANDLE_NULL( data );
    for (int i=0; i<size; i++)
        data[i] = rand();

    return data;
}

// ffcoefs initialization replacement kernel
__global__ void ffcoefs_init_better(Matrix device, int e1)
{
   //int i = blockIdx.x;
   //int j = threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int e2 = e1 * 2;
	unsigned tw = -e1 + i;
	unsigned th = -e1 + j;
	unsigned R = e2;
	double cw = cos(M_PI * tw/(double)R);
	double ch = cos(M_PI * th/(double)R);
	device.elements[i * device.edge + j] = cw*ch;
	//device.elements[i * device.edge + j] = i + 0.1;
}

// ffcoefs initialization replacement kernel
__global__ void ffcoefs_init(double** ffcoefs1, int e1)
{
   //int i = blockIdx.x;
   //int j = threadIdx.x;

	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int e2 = e1 * 2;
	unsigned tw = -e1 + i;
	unsigned th = -e1 + j;
	unsigned R = e2;
	double cw = cos(M_PI * tw/(double)R);
	double ch = cos(M_PI * th/(double)R);
	ffcoefs1[i][j] = cw*ch;


	//for(int i=0; i<EDGE_2; i++)
	//{
	//	ffcoefs[i] = new double[EDGE_2]; 
	//	for(int j=0; j<EDGE_2; j++)
	//	{
	//		unsigned tw = -EDGE_1 + i;
	//		unsigned th = -EDGE_1 + j;
	//		unsigned R = EDGE_2;
	//		double cw = cos( M_PI * tw/(double)R );
	//		double ch = cos( M_PI * th/(double)R );
	//		ffcoefs[i][j] = cw*ch;
	//	}
	//}


   //...code that uses i and j....

}

// Kernel definition 
// newloop<<<gridStuff, blkStuff>>>(screenCUDA, cudaMap.keys, cudaMap.data);
__global__ void newloop(fakeMap cudaMap, unsigned char *screenCUDA, Matrix device, int e1)
{   
	unsigned char r,g,b;
	unsigned clr;
	int resH = 768;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int w = i + 128;
	int h = j + 128;
	int rowLength = e1 * 2;
	int b1, b2, b3, b4;
	b1 = blockIdx.x;
	b2 = blockIdx.y;
	b3 = threadIdx.x;
	b4 = threadIdx.y;

	//for(w=128;w<(128+512);w++)
	//for(h=128;h<(128+512);h++)
	//{ 
	//	b = screen[ 3*(w*resH+h) +0];
	//	g = screen[ 3*(w*resH+h) +1];
	//	r = screen[ 3*(w*resH+h) +2];
	//	clr = ((unsigned)r)+((unsigned)g<<8)+((unsigned)b<<16);

	//	screen[ 3*(w*resH+h) +0] = (unsigned char)(b*ffcoefs[w-128][h-128]);
	//	screen[ 3*(w*resH+h) +1] = (unsigned char)(g*ffcoefs[w-128][h-128]);
	//	screen[ 3*(w*resH+h) +2] = (unsigned char)(r*ffcoefs[w-128][h-128]);

	//	(*ffmap2)[clr] += ffcoefs[w-128][h-128];
	//}


// Kommented out on 2012-09-26 temporarily
	//map<unsigned,double> *ffmap = new map<unsigned,double>();
	int base = 3 * (w * resH + h);
	int twoToOne = i * rowLength + j;
	//if(b1 == 0 && b2 == 0 && b3 == 0 && b4 == 0) {
		//cuPrintf("Screencuda blkX,blkY,thdX,thdY [%d,%d,%d,%d]: %d\n", b1, b2, b3, b4, screenCUDA[base]);
	b = screenCUDA[ base +0];
	g = screenCUDA[ base +1];
	r = screenCUDA[ base +2];
	clr = ((unsigned)r)+((unsigned)g<<8)+((unsigned)b<<16);
	double base2 = device.elements[ twoToOne ];
	//if(b1 == 0 && b2 == 0 && b3 == 0 && b4 == 0) {
	//	cuPrintf("Screencuda blkX,blkY,thdX,thdY [%d,%d,%d,%d]: %d\n", b1, b2, b3, b4, device.elements[twoToOne]);
	//}
	//if(base2 > 0.0001) {
		//cuPrintf("Device is: %f\n", base2);
	//}
	screenCUDA[base +0] = (unsigned char)(b * base2);
	screenCUDA[base +1] = (unsigned char)(g * base2);
	screenCUDA[base +2] = (unsigned char)(r * base2);
	cudaMap.keys[twoToOne] = clr;
	cudaMap.data[twoToOne] = base2;
	//if(b1 == 0 && b2 == 0 && b3 == 0 && b4 == 0)
		//cuPrintf("Screencuda[%d]: %d\n", i, screenCUDA[base]);
	//}
	////screen[ 3*(w*resH+h) +0] = (unsigned char)(b*ffcoefs[w-128][h-128]);
	////screen[ 3*(w*resH+h) +1] = (unsigned char)(g*ffcoefs[w-128][h-128]);
	////screen[ 3*(w*resH+h) +2] = (unsigned char)(r*ffcoefs[w-128][h-128]);
	//
	//((*ffmap)[clr]) += /*1*/ffcoefs1[w-128][h-128];


	//int i = threadIdx.x;
	//int j = threadIdx.y;
	//int scrSize = w * h * 3;
	////scra1 = hipMalloc(
	//scra1[i][j] = 2;/*A[i][j] + B[i][j];*/
}


dim3 block(32, 16);
dim3 grid(32, 16);

//kernel<<<grids, blocks, 1>>>()


/**
* @param destPatch Index of destination patch.
* @param cacheLine Pointer to target cache line.
*/
void FormFactorEngine::fillCacheLine(int destPatch, PatchCacheLine *cacheLine)
{
	PatchRandomAccessEnumerator &patchSet = *patchEnumerator_;

	renderFullScene(destPatch);
	map<unsigned,double> *ffmap = getFF();

	//float S = 256.0f*256.0f+4*256.0f*128.0f;
	float S = 196608.0f;
	
	map<unsigned,double>::iterator iter;
	for(iter = ffmap->begin(); iter != ffmap->end(); iter++ ) {
		if(iter->first != 0xffffff)
		{
				cacheLine->addPatch(iter->first, 2.0*iter->second/S);
			// this condition is due to not offscreen rendering!!!
			if(static_cast<unsigned>(patchSet.count()) > iter->first)
				cacheLine->addPatch(iter->first, 2.0*iter->second/S);
		}
	}
	
	delete ffmap;
}

map<unsigned,double> *FormFactorEngine::getFF()
{
	printf("Now in getFF.\n");
	//map<unsigned,double> *ffmap = new map<unsigned,double>();

	// We can't create a map in CUDA space, so let's create two arrays and use them as a pseudo-map.
	// One will be an unsigned array and one will be a double array.
	//unsigned *map1 = new unsigned[size];
	//double *map2 = new double[size];

	fakeMap cudaMap, localMap;

	//map<unsigned,double> *ffmap_cuda;

	unsigned resW = FormFactorEngine::EDGE_LENGTH;
	unsigned resH = FormFactorEngine::EDGE_LENGTH;

	int screenDataSize = resW * resH * 3;

	//map<unsigned,double> *dev1;
	localMap.keys = (unsigned*) malloc(sizeof(unsigned) * EDGE_2 * EDGE_2);
	localMap.data = (double*) malloc(sizeof(double) * EDGE_2 * EDGE_2);
	HANDLE_ERROR( hipMalloc( &cudaMap.keys, sizeof(unsigned) * EDGE_2 * EDGE_2) );
	HANDLE_ERROR( hipMalloc( &cudaMap.data, sizeof(double) * EDGE_2 * EDGE_2) );
	
	//unsigned resW = 1536;
	//unsigned resH = 1536;
	
	unsigned char *screen = new unsigned char[screenDataSize];
	unsigned char *screenCUDA;
	unsigned char *screenHost = new unsigned char[screenDataSize];

	unsigned w, h, clr;

	unsigned char r,g,b;

	glReadPixels(0, 0, resW, resH, GL_BGR, GL_UNSIGNED_BYTE, screen); 

	// CUDAIZE STARTING HERE
	HANDLE_ERROR( hipMalloc( &screenCUDA, sizeof(unsigned char) * screenDataSize ) );

	HANDLE_ERROR( hipMemcpy( screenCUDA, screen, sizeof(unsigned char) * screenDataSize, hipMemcpyHostToDevice) );
	dim3 gridStuff(16, 16);
	dim3 blkStuff(32, 32);
	// void newloop(fakeMap cudaMap, unsigned char *screenCUDA, Matrix device, int e1)
	//cudaPrintfInit();
	newloop<<<blkStuff, gridStuff>>>(cudaMap, screenCUDA, device, EDGE_1);
	//cudaPrintfDisplay(stdout, true);
	//cudaPrintfEnd();
	HANDLE_ERROR(hipMemcpy( localMap.keys, cudaMap.keys, sizeof(unsigned) * EDGE_2 * EDGE_2, hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy( localMap.data, cudaMap.data, sizeof(double) * EDGE_2 * EDGE_2, hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy( screenHost, screenCUDA, sizeof(unsigned char) * screenDataSize, hipMemcpyDeviceToHost));

	//for(int a = 0; a < screenDataSize; a++) {
	//	if(screen[a] != screenHost[a]) {
	//		printf("Screen[%d]: %d. Cuda: %d.\n", a, screen[a], screenHost[a]);
	//	}
	//}

	//std::cout << "Done with CUDA memcpy!" << endl;
	// After kernel loopa executes, the fakeMap will have the 

	map<unsigned,double> *ffmap = new map<unsigned,double>();
	map<unsigned,double> *ffmap2 = new map<unsigned,double>();
	for(int i = 0; i < EDGE_2 * EDGE_2; i++) {
		(*ffmap)[localMap.keys[i]] += localMap.data[i];
		//if(i%10000 == 0)printf("Localmap key %d, + %f, now %f.\n", localMap.keys[i], localMap.data[i], (*ffmap)[localMap.keys[i]]);
	}

	//int w, h;
	//for(w=128;w<(128+512);w++)
	//	for(h=128;h<(128+512);h++)
	//	{ 
	//		b = screen[ 3*(w*resH+h) +0];
	//		g = screen[ 3*(w*resH+h) +1];
	//		r = screen[ 3*(w*resH+h) +2];
	//		clr = ((unsigned)r)+((unsigned)g<<8)+((unsigned)b<<16);

	//		screen[ 3*(w*resH+h) +0] = (unsigned char)(b*ffcoefs[w-128][h-128]);
	//		screen[ 3*(w*resH+h) +1] = (unsigned char)(g*ffcoefs[w-128][h-128]);
	//		screen[ 3*(w*resH+h) +2] = (unsigned char)(r*ffcoefs[w-128][h-128]);

	//		((*ffmap2)[clr]) += ffcoefs[w-128][h-128];
	//	}

		//map<unsigned,double>::iterator it;
		////for(int i = 0; i < ffmap2.size(); i++) {
		//int i = 0;
		//printf("ffmap2 size: %d.  ffmap size: %d.\n", ffmap2->size(), ffmap->size());
		//for(it = ffmap2->begin(); it != ffmap2->end(); it++) {
		//	unsigned key = it->first;
		//	double dat = it->second;
		//	//if((ffmap2[i] - (*ffmap)[i]) > EPSILON) {
		//	if(((*ffmap2)[key] - (*ffmap)[key]) > EPSILON) {
		//		printf("%05d: CUDA ffmap[%d]: %f.  Real: %f. Iterator: %f.\n", i, key, (*ffmap)[key], (*ffmap2)[key], dat);
		//	}
		//	i++;
		//}
	//HANDLE_ERROR( hipMemcpy( (void*)ffmap, (const void*)ffmap_cuda, sizeof(map<unsigned,double>) * sizeof(ffmap_cuda), hipMemcpyDeviceToHost) );

	//HANDLE_ERROR( hipMemcpy( (void*)screenCUDA, (const void*)screen, sizeoYeahf(unsigned char) * sizeof(screen), hipMemcpyHostToDevice) );


		//glDrawPixels(resW, resH, GL_BGR, GL_UNSIGNED_BYTE, screen);

	//hipFree(screenCUDA);
	//hipFree(device.elements);
	//hipFree(cudaMap.keys);
	//hipFree(cudaMap.data);
	delete[] screen;
	//delete[] localMap.keys;
	//delete[] localMap.data;
	//delete &cudaMap;
	//delete &localMap;
	//delete screenHost;
	//delete screenCUDA;

	return ffmap;

}

__global__ void addSomething(int a, int b, int *c) {
	*c = a + b;
}

/**
 * @param  patchIterator
 */
FormFactorEngine::FormFactorEngine (PatchRandomAccessEnumerator *patchEnumerator):
	patchEnumerator_(patchEnumerator)
{
	createGLWindow();
	//ffcoefs = new double*[EDGE_2];
	ffcoefs = (double**)malloc(sizeof(double*) * EDGE_2);
	//double** ffcoefs_cuda;
	size_t pitch;
	int n;
	int *dev_n;
	std::cout << "Starting some CUDA stuff!" << endl;
	double* devPtr;
	//Matrix host, device;
	host.edge = EDGE_2;
	device.edge = EDGE_2;
	size_t size = EDGE_2 * EDGE_2 * sizeof(double);

	//HANDLE_ERROR( hipMalloc( (void**)&dev_n, sizeof(int) ) );
	//HANDLE_ERROR( hipMalloc( (void**)&ffcoefs_cuda, sizeof(double*) * EDGE_2 * EDGE_2 ) );
	//HANDLE_ERROR( hipMallocPitch( &devPtr, &pitch,sizeof(double)*EDGE_2, EDGE_2) );
	HANDLE_ERROR( hipMalloc( &device.elements, size ));

	host.elements = (double*)malloc(size);

	std::cout << "Malloc'ed some CUDA stuff (and a local array, hooray)! Calling a loop!" << endl;

	/*for(int i = 0; i < EDGE_2; i++)
		ffcoefs[i] = new double[EDGE_2];*/
	dim3 initgrid(32,32);
	dim3 initblk(16,16);
	//ffcoefs_init<<<initgrid, initblk>>>(ffcoefs_cuda, EDGE_1);
	//ffcoefs_init_better<<<initblk, initgrid>>>(device, EDGE_1);
	ffcoefs_init_better<<<initgrid, initblk>>>(device, EDGE_1);
	//addSomething<<<1,1>>>(2, 7, dev_n);

	std::cout << "Done looping, now we're copying back!" << endl;

	HANDLE_ERROR( hipMemcpy( host.elements, device.elements, size, hipMemcpyDeviceToHost) );
	//HANDLE_ERROR( hipMemcpy( &ffcoefs, ffcoefs_cuda, sizeof(double*) * EDGE_2 * EDGE_2, hipMemcpyDeviceToHost) );
	//HANDLE_ERROR( hipMemcpy( &n, dev_n, sizeof(int), hipMemcpyDeviceToHost) );

	//std::cout << "Done with hipMemcpy!" << endl;
	 
	//hipFree(device.elements);
	//printf("Hey, look, 2+7=%d!\n(That just indicates excitement, not factorial. [For Patrick])\n", n);
	//hipFree(dev_n);

	//std::cout << "Done with hipFree!" << endl;

	//double** ffcoefs_local = (double**) malloc(sizeof(double*) * EDGE_2);

	//for(int i=0; i<EDGE_2; i++)
	//{
	//	ffcoefs_local[i] = new double[EDGE_2]; 
	//	for(int j=0; j<EDGE_2; j++)
	//	{
	//		unsigned tw = -EDGE_1 + i;
	//		unsigned th = -EDGE_1 + j;
	//		unsigned R = EDGE_2;
	//		double cw = cos( M_PI * tw/(double)R );
	//		double ch = cos( M_PI * th/(double)R );
	//		ffcoefs_local[i][j] = cw*ch;
	//	}
	//}

	//ffcoefs = ffcoefs_local;
	
	//for(int i=0; i < EDGE_2; i++) {
	//	ffcoefs[i] = new double[EDGE_2];
	//	for(int j=0; j < EDGE_2; j++) {
	//		ffcoefs[i][j] = host.elements[i*EDGE_2+j];
	//		if((ffcoefs[i][j] - ffcoefs_local[i][j] > EPSILON) || ((i%100+j== 34))) // && (i % 50 == 0))
	//			printf("Cuda [%d, %d]: %f. Real: %f.\n", i, j, ffcoefs[i][j], ffcoefs_local[i][j]);
	//	}
	//}

	std::cout << "ffcoefs is now full of good stuff." << endl;
	//HANDLE_ERROR( hipMemcpy( (void*)screenCUDA, (const void*)screen, sizeof(unsigned char) * sizeof(screen), hipMemcpyHostToDevice) );

	//for(int i=0; i<EDGE_2; i++)
	//{
	//	ffcoefs[i] = new double[EDGE_2]; 
	//	for(int j=0; j<EDGE_2; j++)
	//	{
	//		unsigned tw = -EDGE_1 + i;
	//		unsigned th = -EDGE_1 + j;
	//		unsigned R = EDGE_2;
	//		double cw = cos( M_PI * tw/(double)R );
	//		double ch = cos( M_PI * th/(double)R );
	//		ffcoefs[i][j] = cw*ch;
	//	}
	//}
}

FormFactorEngine::~FormFactorEngine()
{
#if defined(__WIN32__) || defined(_WIN32) || defined(__CYGWIN__) 
#else
	XCloseDisplay(dpy);
#endif
	for(int i=0; i<EDGE_2; i++)
		delete[] ffcoefs[i];
	delete[] ffcoefs;
}

#if defined(__WIN32__) || defined(_WIN32) || defined(__CYGWIN__)
LRESULT CALLBACK WndProc(HWND hWnd, UINT message, WPARAM wParam, LPARAM lParam)
{
	switch (message) 
	{
		default:
			return DefWindowProc(hWnd, message, wParam, lParam);
	}
	return 0;
}
#else
#endif

void FormFactorEngine::createGLWindow()
{
#if defined(__WIN32__) || defined(_WIN32) || defined(__CYGWIN__)
	HINSTANCE hInstance = NULL;

	WNDCLASSEX wcex;
	
	wcex.cbSize = sizeof(WNDCLASSEX); 
	wcex.style= CS_HREDRAW | CS_VREDRAW;
	wcex.lpfnWndProc= (WNDPROC)WndProc;
	wcex.cbClsExtra= 0;
	wcex.cbWndExtra= 0;
	wcex.hInstance= hInstance;
	wcex.hIcon= 0;
	wcex.hCursor= LoadCursor(NULL, IDC_ARROW);
	wcex.hbrBackground= (HBRUSH)(COLOR_WINDOW+1);
	wcex.lpszMenuName= 0;
	wcex.lpszClassName= "RadiosityRenderWindowClass";
	wcex.hIconSm= 0;
	
	RegisterClassEx(&wcex);
	
	HWND hWnd = CreateWindow("RadiosityRenderWindowClass", "Radiosity render window", 
		WS_POPUP, 0, 0, EDGE_LENGTH, EDGE_LENGTH, NULL, NULL, hInstance, NULL);
	
	HDC hDC = GetDC( hWnd );
	HGLRC  hglrc;
	
    PIXELFORMATDESCRIPTOR pfd;
    int iFormat;
    ZeroMemory( &pfd, sizeof( pfd ) );
    pfd.nSize = sizeof( pfd );
    pfd.nVersion = 1;
    pfd.dwFlags = PFD_DRAW_TO_WINDOW | PFD_SUPPORT_OPENGL |
                  PFD_DOUBLEBUFFER;
    pfd.iPixelType = PFD_TYPE_RGBA;
    pfd.cColorBits = 24;
    pfd.cDepthBits = 16;
    pfd.iLayerType = PFD_MAIN_PLANE;
    iFormat = ChoosePixelFormat( hDC, &pfd );
    SetPixelFormat( hDC, iFormat, &pfd );
	
	hglrc = wglCreateContext (hDC);
	wglMakeCurrent (hDC, hglrc);
	
	//ShowWindow(hWnd, SW_SHOW);
	//UpdateWindow(hWnd);
#else
	int attributeList[]={
		GLX_RGBA, GLX_RED_SIZE, 8, GLX_GREEN_SIZE, 8,
		GLX_BLUE_SIZE, 8, GLX_DOUBLEBUFFER, GLX_DEPTH_SIZE, 16,
		None};

	dpy = XOpenDisplay(getenv("DISPLAY"));
	if(!dpy)
	{
		cerr << "Cannot open display!" << endl;
		exit(-1);
	}

	int screen;
	screen = DefaultScreen(dpy);

	XVisualInfo *vi;
	vi = glXChooseVisual(dpy, screen, attributeList);
	if(!vi)
	{
		cerr << "Cannot find visual with desired attributes!" << endl;
		exit(-1);
	}

	GLXContext cx;
	cx = glXCreateContext(dpy, vi, NULL, GL_TRUE);
	if(!cx)
	{
		cerr << "Cannot create context!" << endl;
		exit(-1);
	}

	win = XCreateSimpleWindow(dpy, RootWindow(dpy, vi->screen), 0, 0, EDGE_LENGTH,  EDGE_LENGTH, 0, 0, 0);

	XFree(vi);

	XMapWindow(dpy, win);

	XStoreName(dpy, win, "Computing form factors...");

	XSync(dpy,false);

	XEvent ev;
        memset(&ev,0,sizeof(ev)); 
        ev.type=ClientMessage; 
        ev.xclient.type = ClientMessage; 
        ev.xclient.message_type = XInternAtom(dpy, "_NET_WM_STATE", 0); 
        ev.xclient.display=dpy; 
        ev.xclient.window=win; 
        ev.xclient.format=32; 
        ev.xclient.data.l[0]= 1; 
        ev.xclient.data.l[1]=XInternAtom(dpy, "_NET_WM_STATE_STAYS_ON_TOP", 0);
        XLockDisplay(dpy); 
        XSendEvent(dpy, XDefaultRootWindow(dpy), 0, SubstructureRedirectMask | SubstructureNotifyMask, &ev); 

        ev.xclient.data.l[1]=XInternAtom(dpy, "_NET_WM_STATE_ABOVE", 0);
        XSendEvent(dpy, XDefaultRootWindow(dpy), 0, SubstructureRedirectMask | SubstructureNotifyMask, &ev); 
        XUnlockDisplay(dpy);

	XSync(dpy,false);

	glXMakeCurrent(dpy, win, cx);
#endif
	// this should be in some event ;-)
	glClearColor(1.0, 1.0, 1.0, 0.0);
	glClearDepth(1.0f);
	glEnable(GL_DEPTH_TEST);
	glDepthFunc(GL_LESS);

	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	glViewport(0, 0, EDGE_LENGTH, EDGE_LENGTH);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluPerspective((double)90.0, (double)EDGE_LENGTH/(double)EDGE_LENGTH, (double)1e-3, (double)50.0);
	glMatrixMode(GL_MODELVIEW);
}

class Vector;
inline float operator* (const Vector &u, const Vector &v);

#ifndef NDEBUG
inline std::ostream& operator<< (std::ostream &out, const Vector &vect);
#endif
	
class Vector {
	public:
		// tohle by mohlo byt pozdeji private
		float dx,dy,dz;

	public:
		static Vector fromTriangle(Triangle &t) {
			return Vector(
					t.vertex[0],
		 			t.vertex[1],
					t.vertex[2]
					);
		}
		
		static float cos(const Vector &u, const Vector &v) {
			return (u*v)/(u.size()*v.size());
		}
		
		float& operator[](int i) {
			if(0==i)
				return dx;
			if(1==i)
				return dy;
			if(2==i)
				return dz;
			return(dx);
		}
		
		const float& operator[](int i) const {
			if(0==i)
				return dx;
			if(1==i)
				return dy;
			if(2==i)
				return dz;
			return(dx);
		}
		
		Vector(const float dx, const float dy, const float dz)
		{
			this->dx = dx;
			this->dy = dy;
			this->dz = dz;
		}

		Vector(const Vertex &a, const Vertex &b) {
			dx = a.x - b.x;
			dy = a.y - b.y;
			dz = a.z - b.z;
		}

		Vector(const Vector &u, const Vector &v) { // normalovy
			dx = u[1]*v[2] - v[1]*u[2];
			dy = u[2]*v[0] - v[2]*u[0];
			dz = u[0]*v[1] - v[0]*u[1];
		}
		
		Vector(const Vertex &a, const Vertex &b, const Vertex &c) {
			Vector u(b,a);
			Vector v(c,a);
			dx = u[1]*v[2] - v[1]*u[2];
			dy = u[2]*v[0] - v[2]*u[0];
			dz = u[0]*v[1] - v[0]*u[1];
		}
		
		float size() const {
			return sqrtf( dx*dx + dy*dy + dz*dz );
		}
/*
		Vertex operator+ (const Vertex &v) {
			return Vertex(v.x+dx, v.y+dy, v.z+dz);
		}
*/
};

inline float operator* (const Vector &u, const Vector &v) {
	return u[0]*v[0] + u[1]*v[1] + u[2]*v[2];
}
#ifndef NDEBUG
inline std::ostream& operator<< (std::ostream &out, const Vector &vect) {
	out << "vect(" <<
			vect[0] << ", " <<
			vect[1] << ", " <<
			vect[2] << ")";
	return out;
}
#endif

inline Vertex operator+ (const Vertex &v, const Vector &n) {
	return Vertex(v.x+n.dx, v.y+n.dy, v.z+n.dz);
}

void FormFactorEngine::drawScene()
{
	PatchRandomAccessEnumerator &patchSet = *patchEnumerator_;
	glBegin(GL_TRIANGLES);
	for(unsigned i=0;i< static_cast<unsigned>(patchSet.count());i++)
	{
		Triangle &t = patchSet[i];
		glColor3ub((i), (i>>8),(i>>16));
		for(int j=0;j<3;j++)
			glVertex3f(t.vertex[j].x,t.vertex[j].y,t.vertex[j].z);
	}
	glEnd();
}

/**
* @param eye Position of camera
* @param center Direction of camera
* @param up Up vector of camera
*/
void gluLookAtV(const Vertex &eye, const Vertex &center, const Vector &up)
{
	gluLookAt(eye.x, eye.y, eye.z, center.x, center.y, center.z, up.dx, up.dy, up.dz);
}

/**
* @param x Offset from left
* @param y Offset from bottom
* @param c Position of camera
* @param at Direction of camera
* @param up Up vector of camera
*/
void FormFactorEngine::renderViewport(const GLint x, const GLint y, const Vertex &c, const Vertex &at, const Vector &up)
{
	glViewport(x,y, 256,256);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluPerspective(90, (double)EDGE_LENGTH/(double)EDGE_LENGTH, 1e-3, 50);
	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
	gluLookAtV( c, at, up );
	drawScene();
}

/**
* @param dest Index of destination patch.
*/
void FormFactorEngine::renderFullScene(int dest)
{
	PatchRandomAccessEnumerator &patchSet = *patchEnumerator_;

	// clear window
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	// destination triangle
	Triangle &t0 = patchSet[dest];

	// center of this triangle
	Vertex c = Triangle::centerOf(t0);
	
	// normal vector and inverse normal vector of this triangle
	Vector norm = Vector::fromTriangle(t0);
	Vector norm_m(-norm.dx,-norm.dy,-norm.dz);
	
	Vector side(norm, Vector(1,2,3));
	if(0 == side.size())
	    side = Vector(norm, Vector(1,1,1)); // a neudelat Vector::operator= ??
	Vector side_m(-side.dx,-side.dy,-side.dz);
	
	// side vectors
	Vector vctD(/*t0.vertex[0], t0.vertex[1]*/side);
	Vector vctC(/*t0.vertex[1], t0.vertex[0]*/side_m);
	Vector vctA( vctD, norm );
	Vector vctB( vctC, norm );
	
	// points for directions of camera (top and 4 side)
	Vertex at = c+norm;
	Vertex atA = c+vctA;
	Vertex atB = c+vctB;
	Vertex atC = c+vctC;
	Vertex atD = c+vctD;
	
	// top
	renderViewport(256, 256, c, at, vctA);
	
	// 1. side
	renderViewport(256, 512, c, atA, norm_m);
	
	// opposite side
	renderViewport(256, 0, c, atB, norm);
	
	// left side
	renderViewport(0, 256, c, atC, vctA);
	
	// right side
	renderViewport(512, 256, c, atD, vctA);
	
	// rendercc
	glFlush();

	// only due to doublebuffering
#if defined(__WIN32__) || defined(_WIN32) || defined(__CYGWIN__) 
#else
	glXSwapBuffers( dpy, win );
#endif
}
